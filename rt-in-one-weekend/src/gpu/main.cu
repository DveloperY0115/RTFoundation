#include "hip/hip_runtime.h"
//
// Created by dveloperY0115 on 1/8/2021.
//

#include "rtweekend-gpu.hpp"
#include "vector3.hpp"
#include "ray.hpp"
#include "sphere.hpp"
#include "moving_sphere.hpp"
#include "hittable_list.hpp"
#include "camera.hpp"

#include "material.hpp"
#include "lambertian.hpp"
#include "metal.hpp"
#include "dielectric.hpp"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // reset device before terminating
        hipDeviceReset();
        exit(99);
    }
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(hittable **d_list, hittable **d_world, camera **d_camera, int nx, int ny, hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;

        // earth
        d_list[0] = new sphere(vector3(0,-1000.0,-1), 1000,
                               new lambertian(color(0.5, 0.5, 0.5)));

        // randomly generated 22 * 22 spheres
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vector3 center(a+RND,0.2,b+RND);
                if(choose_mat < 0.8f) {
                    // diffuse
                    vector3 center2 = center + vector3(0, RND, 0);
                    d_list[i++] = new moving_sphere(center, center2, 0.0, 1.0, 0.2,
                                             new lambertian(color(RND*RND, RND*RND, RND*RND)));
                }
                else if(choose_mat < 0.95f) {
                    // metal
                    d_list[i++] = new sphere(center, 0.2,
                                             new metal(color(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
                }
                else {
                    // dielectric
                    d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }

        // three spheres
        d_list[i++] = new sphere(vector3(0, 1,0),  1.0, new dielectric(1.5));
        d_list[i++] = new sphere(vector3(-4, 1, 0), 1.0, new lambertian(color(0.4, 0.2, 0.1)));
        d_list[i++] = new sphere(vector3(4, 1, 0),  1.0, new metal(color(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world  = new hittable_list(d_list, 22 * 22 + 1 +3);

        vector3 lookfrom(13,2,3);
        vector3 lookat(0,0,0);
        float dist_to_focus = 10.0; (lookfrom-lookat).length();
        float aperture = 0.1;
        float shutter_open_at = 0.0;
        float shutter_close_at = 1.0;
        *d_camera   = new camera(lookfrom,
                                 lookat,
                                 vector3(0,1,0),
                                 30.0,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus,
                                 shutter_open_at,
                                 shutter_close_at
                                 );
    }
}

__global__ void free_world(hittable **d_list, hittable **d_world, camera **d_camera) {
    for(int i=0; i < 22*22+1+3; i++) {
        delete ((sphere *)d_list[i])->mat_ptr;
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__device__ vector3 ray_color(const ray& r, hittable **world, hiprandState *local_rand_state) {

    ray cur_ray = r;
    vector3 cur_attenuation = vector3(1.0, 1.0, 1.0);

    for (int i = 0; i < 50; i++) {
        hit_record rec;

        // while ray keeps bouncing off from objects, iterate
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vector3 attenuation;

            // if the material scatters the ray, track it
            if (rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            } else {
                // otherwise, it's a black body (an object which absorbs all lights)
                return vector3(0.0, 0.0, 0.0);
            }
        } else {
            // ray hit nothing. it's from the background, not an object in the scene
            vector3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f * (unit_direction.y() + 1.0f);
            vector3 color = (1.0f - t) * vector3(1.0, 1.0, 1.0) + t * vector3(0.5, 0.7, 1.0);
            return cur_attenuation * color;
        }
    }
    return vector3(0.0, 0.0, 0.0);  // exceeded recursion
}

//! \brief Initialize CUDA random state
//!
//! \param rand_state a hiprandState variable
__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

//! \brief Initialize rendering system
//!
//! This function initializes random states allocated for each pixel on frame buffer
//! \param max_x width of the image
//! \param max_y height of the image
//! \param rand_state an array of hiprandState variables
__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x >= max_x) || (y >= max_y)) return;

    unsigned int pixel_index = y * max_x + x;

    // each thread gets same seed, a different sequence number
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vector3* fb, int max_x, int max_y, int num_samples,
                       camera** cam, hittable **world, hiprandState* rand_state) {
    // get global pixel coordinate
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x >= max_x) || (y >= max_y)) return;   // don't render outside the image
    unsigned int pixel_index = y * max_x + x;
    hiprandState local_rand_state = rand_state[pixel_index];
    vector3 pixel_color = color(0, 0, 0);
    for (int sample_idx = 0; sample_idx < num_samples; sample_idx++) {
        float u = float(x + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(y + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        pixel_color += ray_color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    pixel_color /= float(num_samples);
    // gamma correction
    pixel_color[0] = sqrt(pixel_color[0]);
    pixel_color[1] = sqrt(pixel_color[1]);
    pixel_color[2] = sqrt(pixel_color[2]);

    fb[pixel_index] = pixel_color;
}

int main() {

    // Image
    const float aspect_ratio = 16.0 / 9.0;
    const int image_width = 1600;
    const int image_height = static_cast<int>(image_width / aspect_ratio);
    static const int num_samples = 100;

    // allocate Frame Buffer for rendering
    int num_pixels = image_width * image_height;
    size_t fb_size = num_pixels * sizeof(vector3);
    vector3 *fb;
    checkCudaErrors(hipMallocManaged((void**) &fb, fb_size));

    // set random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**) &d_rand_state, num_pixels * sizeof(hiprandState)));

    // second random state variable for random world generation
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**) &d_rand_state2, sizeof(hiprandState)));
    rand_init<<<1, 1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // set world and camera
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    hittable **d_list;
    int num_hittables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void**) &d_list, num_hittables * sizeof(hittable*)));
    hittable **d_world;
    checkCudaErrors(hipMalloc((void **) &d_world, sizeof(hittable*)));
    create_world<<<1, 1>>>(d_list, d_world, d_camera, image_width, image_height, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // set dimensions of grid
    int tx = 8;
    int ty = 8;
    dim3 blocks(image_width / tx + 1, image_height / ty + 1);
    dim3 threads(tx, ty);

    std::cerr << "Rendering a " << image_width << "x" << image_height << " image with " << num_samples << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    clock_t start, stop;
    start = clock();
    // render
    render_init<<<blocks, threads>>>(image_width, image_height, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, image_width, image_height, num_samples, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;

    std::cerr << "took " << timer_seconds << " seconds.\n";

    // write output to a file
    std::cout << "P3\n" << image_width << " " << image_height << "\n255\n";
    for (int j = image_height - 1; j >= 0; j--) {
        for (int i = 0; i < image_width; i++) {
            size_t pixel_index = j * image_width + i;
            float r = fb[pixel_index].r();
            float g = fb[pixel_index].g();
            float b = fb[pixel_index].b();
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
    return 0;
}